#include "hip/hip_runtime.h"
#include "../headers/reduction.h"
#include "../headers/gpu.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

namespace gpu {

  template <class T, unsigned int blockSize>
  __global__ void reduce(T *g_idata, T *g_odata, unsigned int n) {
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T *sdata = new T;

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    T mySum = (i < n) ? g_idata[i] : 0;

    if (i + blockSize < n) mySum += g_idata[i + blockSize];

    sdata[tid] = mySum;
    cg::sync(cta);

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
      if (tid < s) {
        sdata[tid] = mySum = mySum + sdata[tid + s];
      }

      cg::sync(cta);
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    if (cta.thread_rank() < 32) {
      // Fetch final intermediate sum from 2nd warp
      if (blockSize >= 64) mySum += sdata[tid + 32];
      // Reduce final warp using shuffle
      for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
        mySum += tile32.shfl_down(mySum, offset);
      }
    }

    // write result for this block to global mem
    if (cta.thread_rank() == 0) g_odata[blockIdx.x] = mySum;
  }

  template <class T, unsigned int blockSize>
  __global__ void reduceToMax(T* g_idata, T* g_odata, unsigned int n) {
      // Handle to thread block group
      cg::thread_block cta = cg::this_thread_block();
      extern __shared__ T sdata[];

      // Load data from global memory to shared memory
      unsigned int tid = threadIdx.x;
      unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

      T myMax = (i < n) ? g_idata[i] : 0;
      if (i + blockSize < n) {
          myMax = max(myMax, g_idata[i + blockSize]);
      }

      sdata[tid] = myMax;
      cg::sync(cta);

      // Perform reduction in shared memory
      for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
          if (tid < s) {
              myMax = max(myMax, sdata[tid + s]);
              sdata[tid] = myMax;
          }
          cg::sync(cta);
      }

      cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

      if (cta.thread_rank() < 32) {
          // Fetch final intermediate maximum from 2nd warp
          if (blockSize >= 64) {
              myMax = max(myMax, sdata[tid + 32]);
          }
          // Reduce final warp using shuffle
          for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
              myMax = max(myMax, tile32.shfl_down(myMax, offset));
          }
      }

      // Write result for this block to global memory
      if (cta.thread_rank() == 0) {
          g_odata[blockIdx.x] = myMax;
      }
  }


  template __global__ void reduce<int, 512>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduce<int, 256>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduce<int, 128>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduce<int, 64>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduce<int, 32>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduce<int, 16>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduce<int, 8>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduce<int, 4>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduce<int, 2>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduce<int, 1>(int *g_idata, int *g_odata, unsigned int n);


  template __global__ void reduceToMax<int, 512>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduceToMax<int, 256>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduceToMax<int, 128>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduceToMax<int, 64>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduceToMax<int, 32>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduceToMax<int, 16>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduceToMax<int, 8>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduceToMax<int, 4>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduceToMax<int, 2>(int *g_idata, int *g_odata, unsigned int n);
  template __global__ void reduceToMax<int, 1>(int *g_idata, int *g_odata, unsigned int n);
}

