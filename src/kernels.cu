#include "hip/hip_runtime.h"
#include "../headers/kernels.h"
#include "../headers/common.h"
#include "../headers/reduction.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cuda_device_runtime_api.h>

#include "../headers/hip/hip_runtime_api.h"

#include <iostream>
#include <sys/types.h>


#define MAX_THREADS  256
#define max(a, b) ((a) > (b) ? (a) : (b))

namespace gpu {

  __global__ void conflictMatrixKernel(ushort *conflictMatrix, ushort *adjMatrix, ushort *chromosome, ushort n) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < n && col < n && col > row) {
      conflictMatrix[n*row + col] = (adjMatrix[n*row + col] == 1) && (chromosome[row] == chromosome[col]);
    }
    // threads = n * n-1
    // idx = tid / n + tid + 1
    // conflictMatrix[idx] = (adjMatrix[idx] == 1) && (chromosome[tid / n] == chromosome[tid%n +1]);
  }
  unsigned int nextPow2(unsigned int x) {
        --x;
      x |= x >> 1;
      x |= x >> 2;
      x |= x >> 4;
      x |= x >> 8;
      x |= x >> 16;
      return ++x;
    }

    void getNumBlocksAndThreads(int n, int &blocks, int &threads) {
      threads = (n < MAX_THREADS * 2) ? nextPow2((n + 1) / 2) : MAX_THREADS;
      blocks = (n + (threads * 2 - 1)) / (threads * 2);
    }

    void chooseAndReduce(ushort* d_odata, ushort* d_idata, unsigned int size, int &blocks) {
      int threads;
      getNumBlocksAndThreads(size, blocks, threads);
      dim3 dimBlock(threads, 1, 1);
      dim3 dimGrid(blocks, 1, 1);
      int smemSize = (threads <= 32) ? 2 * threads * sizeof(int) : threads * sizeof(int);
    
      switch (threads) {
          case 512:reduce<ushort, 512><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 256:reduce<ushort, 256><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 128:reduce<ushort, 128><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 64:reduce<ushort, 64><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 32:reduce<ushort, 32><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 16:reduce<ushort, 16><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 8:reduce<ushort, 8><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 4:reduce<ushort, 4><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 2:reduce<ushort, 2><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 1:reduce<ushort, 1><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
      } 
    }
    
    ushort fittest(ushort *chromosome) {
      int penalty = 0;
      size_t bytes = n*n * sizeof(ushort);
      ushort * h_penaltyMatix = (ushort*)malloc(bytes);
      ushort * d_adjMatrix;
      ushort * d_chromosome;
      ushort * d_conflictMatrix;
      ushort * d_result;

      checkCudaErrors(hipMalloc((void**)&d_adjMatrix, bytes));
      checkCudaErrors(hipMalloc((void**)&d_conflictMatrix, bytes));
      checkCudaErrors(hipMalloc((void**)&d_chromosome, n*sizeof(ushort)));
      checkCudaErrors(hipMalloc((void**)&d_result, bytes));
    
      checkCudaErrors(hipMemset(d_conflictMatrix, 0, bytes));
      checkCudaErrors(hipMemset(d_result, 0, bytes));
      checkCudaErrors(hipMemcpy(d_chromosome, chromosome, n*sizeof(ushort), hipMemcpyHostToDevice));
      for (int i=0; i<n; ++i) {
        checkCudaErrors(hipMemcpyAsync(d_adjMatrix + i*n, adj[i], n*sizeof(ushort), hipMemcpyHostToDevice));
      }

      unsigned int blockThreads = (n + 32 - 1) / 32;
      dim3 conflictsGridDim(blockThreads, blockThreads, 1);
      dim3 conflictsThreadsDim(32, 32, 1);
      conflictMatrixKernel<<<conflictsGridDim, conflictsThreadsDim>>>(d_conflictMatrix, d_adjMatrix, d_chromosome, n);
      getLastCudaError("Kernel execution failed");

      int blocks;
      chooseAndReduce(d_result, d_conflictMatrix, n*n, blocks);
      getLastCudaError("Kernel execution failed");

      checkCudaErrors(hipMemcpy(h_penaltyMatix, d_result, bytes, hipMemcpyDeviceToHost));
      for (int i = 0; i < blocks; i++) {
        penalty += h_penaltyMatix[i];
      } 

      checkCudaErrors(hipFree(d_conflictMatrix));
      checkCudaErrors(hipFree(d_adjMatrix));
      checkCudaErrors(hipFree(d_chromosome));
      checkCudaErrors(hipFree(d_result));

      return penalty;
    }

  __global__ void crossoverKernel(ushort* newFirst, ushort* newSecond, ushort* first, ushort* second, ushort a, ushort n) {
      int tid = blockIdx.x * blockDim.x + threadIdx.x;

      if (tid < a) {
          newFirst[tid] = second[tid];
          newSecond[tid] = first[tid];
      } else if (tid < n) {
          newFirst[tid] = first[tid];
          newSecond[tid] = second[tid];
      }
  }

  std::vector<std::vector<int>*> *crossover(std::vector<int> *first, std::vector<int> *second, int n) {
      int a = rand() % (n - 1);
      int size = n * sizeof(ushort);

      // Allocate device memory
      ushort* devFirst;
      ushort* devSecond;
      ushort* devNewFirst;
      ushort* devNewSecond;
      hipMalloc((void**)&devFirst, size);
      hipMalloc((void**)&devSecond, size);
      hipMalloc((void**)&devNewFirst, size);
      hipMalloc((void**)&devNewSecond, size);

      // Copy input data to device memory
      hipMemcpy(devFirst, first->data(), size, hipMemcpyHostToDevice);
      hipMemcpy(devSecond, second->data(), size, hipMemcpyHostToDevice);

      // Launch kernel
      int blockSize = 256;
      int gridSize = (n + blockSize - 1) / blockSize;
      crossoverKernel<<<gridSize, blockSize>>>(devNewFirst, devNewSecond, devFirst, devSecond, a, n);

      // Copy result back to host memory
      std::vector<int>* newFirst = new std::vector<int>(n);
      std::vector<int>* newSecond = new std::vector<int>(n);
      hipMemcpy(newFirst->data(), devNewFirst, size, hipMemcpyDeviceToHost);
      hipMemcpy(newSecond->data(), devNewSecond, size, hipMemcpyDeviceToHost);

      // Free device memory
      hipFree(devFirst);
      hipFree(devSecond);
      hipFree(devNewFirst);
      hipFree(devNewSecond);

      // Create result vector
      std::vector<std::vector<int>*>* res = new std::vector<std::vector<int>*>();
      res->push_back(newFirst);
      res->push_back(newSecond);

      return res;
  }

  void chooseAndReduceToMax(ushort* d_odata, ushort* d_idata,unsigned int size, int &blocks) {
      int  threads;
      getNumBlocksAndThreads(size, blocks, threads);
      dim3 dimBlock(threads, 1, 1);
      dim3 dimGrid(blocks, 1, 1);
      int smemSize = (threads <= 32) ? 2 * threads * sizeof(int) : threads * sizeof(int);
    
      switch (threads) {
          case 512:reduceToMax<ushort, 512><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 256:reduceToMax<ushort, 256><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 128:reduceToMax<ushort, 128><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 64:reduceToMax<ushort, 64><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 32:reduceToMax<ushort, 32><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 16:reduceToMax<ushort, 16><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 8:reduceToMax<ushort, 8><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 4:reduceToMax<ushort, 4><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 2:reduceToMax<ushort, 2><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
          case 1:reduceToMax<ushort, 1><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size); break;
      } 
    }

    ushort reduceToMax(ushort* input, ushort n) {
      // Device variables
      ushort *d_input, *d_output;
      ushort result;

      // Allocate memory on the device
      hipMalloc((void**)&d_input, n * sizeof(ushort));
      hipMalloc((void**)&d_output, n * sizeof(ushort));

      // Copy the input vector from the host to the device
      hipMemcpy(d_input, input, n * sizeof(ushort), hipMemcpyHostToDevice);

      // Determine the block and grid dimensions
      int blockSize = 256;
      int gridSize = (n + blockSize - 1) / blockSize;

      // Perform parallel reduction within each block
      reduceBlockMax<<<gridSize, blockSize, blockSize * sizeof(ushort)>>>(d_input, d_output, n);

      // Perform final reduction across block maximum values
      reduceFinalMax<<<1, blockSize>>>(d_output, gridSize);

      // Copy the final maximum value from the device to the host
      hipMemcpy(&result, d_output, sizeof(ushort), hipMemcpyDeviceToHost);

      // Free the allocated memory on the device
      hipFree(d_input);
      hipFree(d_output);

      return result;
    }

    ushort colorCount(ushort* chromosome) {
      return reduceToMax(chromosome, n); 
    }
  
  //   int colorCount(std::vector<int>* chromosome) {
  //     int bytes = n * sizeof(int);
  //     int* d_data;
  //     int* d_result;
  //     checkCudaErrors(hipMalloc((void**)&d_data, bytes));
  //     checkCudaErrors(hipMalloc((void**)&d_result, bytes));
  //     checkCudaErrors(hipMemcpy(d_data, chromosome->data(), bytes, hipMemcpyHostToDevice));
  //
  //     int blocks;
  //     chooseAndReduceToMax(d_result, d_data, n, blocks);
  //     getLastCudaError("Kernel execution failed");
  // 
  //     int result;
  //     int *partialMaxima = (int*)malloc(bytes);
  //     // checkCudaErrors(hipMemcpy(partialMaxima, d_result, bytes, hipMemcpyDeviceToHost));
  //     // for (int i=0; i< blocks; ++i) {
  //     //   result = max(result, partialMaxima[i]);
  //     // }
  //
  //     // Invoke the reduce kernel
  //     // reduceToMax<int, blockSize><<<numBlocks, blockSize>>>(d_data, d_intermediate, n);
  //
  //     // Invoke the reduce kernel again to obtain the final result
  //     // reduceToMax<int, blockSize><<<1, blockSize>>>(d_intermediate, d_result, numBlocks);
  //
  //     // Copy the result from GPU to CPU
  //
  //     // Clean up GPU memory
  //     checkCudaErrors(hipFree(d_data));
  //     // hipFree(d_intermediate);
  //     checkCudaErrors(hipFree(d_result));
  //     free(partialMaxima);
  //
  //     return result;
  // }


  __global__ void countColorsKernel(ushort* chromosome, ushort* colors, int size) {
      int tid = blockIdx.x * blockDim.x + threadIdx.x;

      if (tid < size) {
          atomicAdd((int*)&colors[chromosome[tid] - 1], 1);
      }
  }

  __global__ void swapColorsKernel(ushort* chromosome, ushort* swapTab, ushort* newChromosome, int size) {
      int tid = blockIdx.x * blockDim.x + threadIdx.x;

      if (tid < size) {
          int color = chromosome[tid] - 1;
          newChromosome[tid] = (swapTab[color] == -1) ? chromosome[tid] : swapTab[color] + 1;
      }
  }

  // std::vector<int>* minimalizeColors(std::vector<int>* chromosome, int maxColors) {
  //       int* d_chromosome;
  //       int* d_colors;
  //       int* d_swapTab;
  //       int* d_newChromosome;

  //       checkCudaErrors(hipMalloc((void**)&d_chromosome, n * sizeof(int)));
  //       checkCudaErrors(hipMalloc((void**)&d_colors, maxColors * sizeof(int)));
  //       checkCudaErrors(hipMalloc((void**)&d_swapTab, maxColors * sizeof(int)));
  //       checkCudaErrors(hipMalloc((void**)&d_newChromosome, n * sizeof(int)));
  //
  //       checkCudaErrors(hipMemcpy(d_chromosome, chromosome->data(), n * sizeof(int), hipMemcpyHostToDevice));
  //       unsigned int blockSizeCount = 256;
  //       unsigned int numBlocksCount = (n + blockSizeCount - 1) / blockSizeCount;
  //
  //       countColorsKernel<<<numBlocksCount, blockSizeCount>>>(d_chromosome, d_colors, n);
  //       getLastCudaError("error invoking kernel");
  //
  //       // Set up grid and block dimensions for swapping colors
  //       unsigned int blockSizeSwap = 256;
  //       unsigned int numBlocksSwap = (n + blockSizeSwap - 1) / blockSizeSwap;
  //
  //       swapColorsKernel<<<numBlocksSwap, blockSizeSwap>>>(d_chromosome, d_swapTab, d_newChromosome, n);
  //       getLastCudaError("error invoking kernel");
  //
  //       std::vector<int>* newChromosome = new std::vector<int>(n);
  //       checkCudaErrors(hipMemcpy(newChromosome->data(), d_newChromosome, n * sizeof(int), hipMemcpyDeviceToHost));
  //
  //       checkCudaErrors(hipFree(d_chromosome));
  //       checkCudaErrors(hipFree(d_colors));
  //       checkCudaErrors(hipFree(d_swapTab));
  //       checkCudaErrors(hipFree(d_newChromosome));
  //
  //       return newChromosome;
  //   }


}
